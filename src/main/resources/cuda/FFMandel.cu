
#include <hip/hip_runtime.h>
#define MODE_MANDEL 1
#define MODE_MANDEL_DISTANCE 2
#define MODE_JULIA 3

#define WIDTH gridDim.x*blockDim.x
#define HEIGHT gridDim.y*blockDim.y
#define X ((blockIdx.x * blockDim.x) + threadIdx.x)
#define Y ((blockIdx.y * blockDim.y) + threadIdx.y)

#define SPLIT  4097.0f // 2^12+1, for IEEE float

__device__ inline float2 mul(const float2 pFF1, const float2 pFF2) {
    const float hi = pFF1.x;
    const float lo = pFF1.y;
    const float yhi = pFF2.x;
    const float ylo = pFF2.y;

    float hx, tx, hy, ty, C, c;
    C = SPLIT * hi;
    hx = C - hi;
    c = SPLIT * yhi;
    hx = C - hx;
    tx = hi - hx;
    hy = c - yhi;
    C = hi * yhi;
    hy = c - hy;
    ty = yhi - hy;
    c = ((((hx * hy - C) + hx * ty) + tx * hy) + tx * ty) + (hi * ylo + lo * yhi);
    const float zhi = C + c;
    hx = C - zhi;
    const float zlo = c + hx;

    return make_float2(zhi, zlo);
}

__device__ inline float2 mulFloat(const float2 pFF1, const float pFloat) {
    const float hi = pFF1.x;
    const float lo = pFF1.y;
    const float yhi = pFloat;

    float hx, tx, hy, ty, C, c;
    C = SPLIT * hi;
    hx = C - hi;
    c = SPLIT * yhi;
    hx = C - hx;
    tx = hi - hx;
    hy = c - yhi;
    C = hi * yhi;
    hy = c - hy;
    ty = yhi - hy;
    c = ((((hx * hy - C) + hx * ty) + tx * hy) + tx * ty) + (lo * yhi);
    const float zhi = C + c;
    hx = C - zhi;
    const float zlo = c + hx;

    return make_float2(zhi, zlo);
}


__device__ inline float2 add(const float2 pFF1, const float2 pFF2) {
    const float hi = pFF1.x;
    const float lo = pFF1.y;
    const float yhi = pFF2.x;
    const float ylo = pFF2.y;

    float H, h, T, t, S, s, e, f;
    S = hi + yhi;
    T = lo + ylo;
    e = S - hi;
    f = T - lo;
    s = S - e;
    t = T - f;
    s = (yhi - e) + (hi - s);
    t = (ylo - f) + (lo - t);
    e = s + T;
    H = S + e;
    h = e + (S - H);
    e = t + h;

    const float zhi = H + e;
    const float zlo = e + (H - zhi);

    return make_float2(zhi, zlo);
}

__device__ inline float2 addFloat(const float2 pFF1, const float y) {
    float hi = pFF1.x;
    float lo = pFF1.y;

    float H, h, S, s, e, f;
    S = hi + y;
    e = S - hi;
    s = S - e;
    s = (y - e) + (hi - s);
    f = s + lo;
    H = S + f;
    h = f + (S - H);
    hi = H + h;
    lo = h + (H - hi);

    return make_float2(hi, lo);
}

__device__ inline float2 sub(const float2 pFF1, const float2 pFF2) {
    return add(pFF1, make_float2(-pFF2.x, -pFF2.y));
}


extern "C"
__global__ void compute(
        int *iters,
        double *lastValuesR,
        double *lastValuesI,
        double *distancesR,
        double *distancesI,
        const int mode,
        const int4 tile,
        const float2 xStart,
        const float2 yStart,
        const float2 juliaCr,
        const float2 juliaCi,
        const float2 xInc,
        const float2 yInc,
        const int maxIterations,
        const double sqrEscapeRadius
) {

    if (X >= tile.z || Y >= tile.w) {      // tile.z is width of tile, tile.w is height of tile
        return;
    }


    const float2 x = add(make_float2(xStart.x, xStart.y), mulFloat(make_float2(xInc.x, xInc.y), X));
    const float2 y = add(make_float2(yStart.x, yStart.y), mulFloat(make_float2(yInc.x, yInc.y), Y));

    const float2 cr = mode == MODE_JULIA ? juliaCr : x;
    const float2 ci = mode == MODE_JULIA ? juliaCi : y;

    const float escape = (float) sqrEscapeRadius;

    float2 zr = x;
    float2 zi = y;

    float2 tmp;

    // distance
    float2 dr = make_float2(1, 0);
    float2 di = make_float2(0, 0);
    float2 new_dr;

    int count = 0;

    for (; count < maxIterations; count++) {
        const float2 zrsqr = mul(zr, zr);
        const float2 zisqr = mul(zi, zi);

        if (add(zrsqr, zisqr).x >= escape) {
            break;
        }

        if (mode == MODE_MANDEL_DISTANCE) {
//            new_dr = 2.0f * (zr * dr - zi * di) + 1.0f;
            new_dr = addFloat(mulFloat(sub(mul(zr, dr), mul(zi, di)), 2.0f), 1.0f);
//            di = 2.0f * (zr * di + zi * dr);
            di = mulFloat(add(mul(zr, di), mul(zi, dr)), 2.0f);
            dr = new_dr;
        }

        tmp = add(sub(zrsqr, zisqr), cr);
        zi = add(mulFloat(mul(zr, zi), 2.0f), ci);
        zr = tmp;

    }
    const int tIndex = X + Y * tile.z;  // tile.z is width of tile
    iters[tIndex] = count;
    lastValuesR[tIndex] = (double) zr.x + (double) zr.y;
    lastValuesI[tIndex] = (double) zi.x + (double) zi.y;
    if (mode == MODE_MANDEL_DISTANCE) {
        distancesR[tIndex] = (double) dr.x + (double) dr.y;
        distancesI[tIndex] = (double) di.x + (double) di.y;
    }
}