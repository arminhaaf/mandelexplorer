
#include <hip/hip_runtime.h>
#define MODE_MANDEL 1
#define MODE_MANDEL_DISTANCE 2
#define MODE_JULIA 3

#define WIDTH gridDim.x*blockDim.x
#define HEIGHT gridDim.y*blockDim.y
#define X ((blockIdx.x * blockDim.x) + threadIdx.x)
#define Y ((blockIdx.y * blockDim.y) + threadIdx.y)

__device__ inline float2 mul(const float2 pFF1, const float2 pFF2) {
    const float hi = pFF1.x;
    const float lo = pFF1.y;
    const float yhi = pFF2.x;
    const float ylo = pFF2.y;

       float t, tau, u, v, w;

           t = hi * yhi;            /* Highest order float term.  */

           if (t == 0) {
               return make_float2(0,0);
           }

           tau = fma(hi, yhi, -t);
           v = hi * ylo;
           w = lo * yhi;
           tau += v + w;        /* Add in other second-order terms.	 */
           u = t + tau;

       return make_float2(u, (t - u) + tau);
}

__device__ inline float2 mulFloat(const float2 pFF1, const float pDouble) {
    const float hi = pFF1.x;
    const float lo = pFF1.y;
    const float yhi = pDouble;

       float t, tau, u, w;

           t = hi * yhi;            /* Highest order float term.  */

           if (t == 0) {
               return make_float2(0,0);
           }

           tau = fma(hi, yhi, -t);
           w = lo * yhi;
           tau += w;        /* Add in other second-order terms.	 */
           u = t + tau;

       return make_float2(u, (t - u) + tau);
}


__device__ inline float2 add(const float2 pFF1, const float2 pFF2) {
    const float hi = pFF1.x;
    const float lo = pFF1.y;
    const float yhi = pFF2.x;
    const float ylo = pFF2.y;

        float z, q, zz, xh;

        z = hi + yhi;

        q = hi - z;
        zz = q + yhi + (hi - (q + z)) + lo + ylo;

        /* Keep -0 result.  */
        if (zz == 0.0) {
            return make_float2(z,0);
        }

        xh = z + zz;

    return make_float2(xh,z - xh + zz);

}

__device__ inline float2 addFloat(const float2 pFF1, const float y) {
    float hi = pFF1.x;
    float lo = pFF1.y;

        float z, q, zz, xh;

        z = hi + y;

        q = hi - z;
        zz = q + y + (hi - (q + z)) + lo;

        /* Keep -0 result.  */
        if (zz == 0.0) {
            return make_float2(z,0);
        }

        xh = z + zz;

    return make_float2(xh,z - xh + zz);
}

__device__ inline float2 sub(const float2 pFF1, const float2 pFF2) {
    return add(pFF1, make_float2(-pFF2.x, -pFF2.y));
}


extern "C"
__global__ void compute(
        int *iters,
        double *lastValuesR,
        double *lastValuesI,
        double *distancesR,
        double *distancesI,
        const int mode,
        const int4 tile,
        const float2 xStart,
        const float2 yStart,
        const float2 juliaCr,
        const float2 juliaCi,
        const float2 xInc,
        const float2 yInc,
        const int maxIterations,
        const double sqrEscapeRadius
) {

    if (X >= tile.z || Y >= tile.w) {      // tile.z is width of tile, tile.w is height of tile
        return;
    }


    const float2 x = add(make_float2(xStart.x, xStart.y), mulFloat(make_float2(xInc.x, xInc.y), X));
    const float2 y = add(make_float2(yStart.x, yStart.y), mulFloat(make_float2(yInc.x, yInc.y), Y));

    const float2 cr = mode == MODE_JULIA ? juliaCr : x;
    const float2 ci = mode == MODE_JULIA ? juliaCi : y;

    const float escape = (float) sqrEscapeRadius;

    float2 zr = x;
    float2 zi = y;

    float2 tmp;

    // distance
    float2 dr = make_float2(1, 0);
    float2 di = make_float2(0, 0);
    float2 new_dr;

    int count = 0;

    for (; count < maxIterations; count++) {
        const float2 zrsqr = mul(zr, zr);
        const float2 zisqr = mul(zi, zi);

        if (add(zrsqr, zisqr).x >= escape) {
            break;
        }

        if (mode == MODE_MANDEL_DISTANCE) {
//            new_dr = 2.0f * (zr * dr - zi * di) + 1.0f;
            new_dr = addFloat(mulFloat(sub(mul(zr, dr), mul(zi, di)), 2.0f), 1.0f);
//            di = 2.0f * (zr * di + zi * dr);
            di = mulFloat(add(mul(zr, di), mul(zi, dr)), 2.0f);
            dr = new_dr;
        }

        tmp = add(sub(zrsqr, zisqr), cr);
        zi = add(mulFloat(mul(zr, zi), 2.0f), ci);
        zr = tmp;

    }
    const int tIndex = X + Y * tile.z;  // tile.z is width of tile
    iters[tIndex] = count;
    lastValuesR[tIndex] = (double) zr.x + (double) zr.y;
    lastValuesI[tIndex] = (double) zi.x + (double) zi.y;
    if (mode == MODE_MANDEL_DISTANCE) {
        distancesR[tIndex] = (double) dr.x + (double) dr.y;
        distancesI[tIndex] = (double) di.x + (double) di.y;
    }
}