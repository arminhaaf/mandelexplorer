
#include <hip/hip_runtime.h>
#include <stdio.h>

#define WIDTH gridDim.x*blockDim.x
#define HEIGHT gridDim.y*blockDim.y
#define X ((blockIdx.x * blockDim.x) + threadIdx.x)
#define Y ((blockIdx.y * blockDim.y) + threadIdx.y)

extern "C"
__global__ void computeFloat(
      int *iters,
      float4 area,
      int maxIterations,
      float sqrEscapeRadius
      ) {

   const float cr = area.x + X*area.z;
   const float ci = area.y + Y*area.w;

   float zr = 0;
   float zi = 0;
   float new_zr = 0.0f;

   int count = 0;
   for (; count<maxIterations; count++){
        new_zr = (zr * zr - zi * zi) + cr;
        zi = ((2.0f * zr) * zi) + ci;
        zr = new_zr;

        if ( (zr * zr + zi * zi) >= sqrEscapeRadius ) {
            break;
        }
   }
   const int tIndex = X + Y * WIDTH;
   iters[tIndex]  = count;
}



int main() {
    int tWidth = 5000;
    int tHeight = 5000;
    int tMaxIter = 1000;

    float4 tArea = {-1.5f,-1.0f, 2.0f/tWidth, 2.0f/tHeight};

    int *tIters = (int*)malloc(tWidth*tHeight*sizeof(int));

    int *cuIters;

    hipMalloc((void **)&cuIters, sizeof(int)*tWidth*tHeight);
    hipMemcpy(cuIters, tIters ,tHeight*tWidth*sizeof(int) ,hipMemcpyHostToDevice);

    int tThreadsX = 8;
    int tThreadsY = 4;

    dim3 blockSize = dim3( tThreadsX,tThreadsY);
    dim3 gridSize  = dim3( tWidth/tThreadsX, tHeight/tThreadsY );

    computeFloat<<<gridSize, blockSize>>>(
          cuIters,
          tArea,
          tMaxIter,
          4);
     hipDeviceSynchronize();

    hipMemcpy(tIters, cuIters ,tHeight*tWidth*sizeof(int) ,hipMemcpyDeviceToHost);

    // 160 chars width and 80 chars height
    for ( int y=0;y<tHeight; y+=tHeight/80) {
        for ( int x=0; x<tWidth; x+=tWidth/160) {
            if ( tIters[x+y*tWidth]==tMaxIter) {
              printf("X");
            } else {
                printf(".");
            }
        }
        printf("\n");
    }

}

